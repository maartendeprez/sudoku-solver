/* -*- mode: c -*- */
/*
 * Sudoku solver
 *
 * Author: Maarten Deprez <deprez.maarten@gmail.com>
 */

#define BRUTEFORCE 1 /* go on guessing if deterministic algorithm
			yields an incompletely defined solution */
/*#define COMB_BUF 1*//* use combination buffer; faster, but uses
		      * a block of memory (128MB for 5x5) per
		      * group resolver; not possible yet with
		      * parallelization */
/*#define CPUEXT 1*/ /* use CPU extensions (doesn't work for GPU) */
/*#define DEBUG_STEP 1*/


#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/device_functions.h>


/* Field type and predefined values
 * unsigned int is enough for 5x5x5
 */

typedef unsigned int field;

typedef struct params {

  int dimension;
  int groupsize;
  int boardsize;
  int allbits;
  const char *symbols;
#if 0
  field *groups;
  int *ms;
#endif
#ifdef COMB_BUF
  field *buf;
#endif
} params_t;

typedef struct sudoku {

  struct params *pars;
  field *board;
  char *path;
  int steps;
  int tries;
  int allsteps;
  int alltries;
  int solutions;
} sudoku_t;


/* Function prototypes */

void setparams(params_t *p, int dimension, const char *syms);

int load(sudoku_t *game, params_t *pars, int pn, char *path);
void unload(sudoku_t *game);

void show(sudoku_t *game);
void showmeta(sudoku_t *game, int s);
void showmeta2(sudoku_t *game);

int solve(sudoku_t *game);
int solve2(sudoku_t *game, int ms);

const char *statusmsg(int s);


static int solve1(params_t *p, field *board);

static __global__ void solvegroup(int boardsize, int groupsize, int allbits);

static void showboard(params_t *p, field *board);
static int sym_to_field(params_t *p, char c, field *b);
static char field_to_sym(params_t *p, field b);

static int status(params_t *p, field *board);
static int verify(params_t *p, field *board);

static __device__ field combination(const field *group, field c);

#ifdef CPUEXT

#include <x86intrin.h>

#define nbits _mm_popcnt_u32
#define fstbit __bsfd
//#define clrbit(b) (c = __blsr_u32(b))

#define clrbit(b) (b &= b - 1)

#else

static inline int nbits(field b);
static inline int fstbit(field b);

#define clrbit(b) (b &= b - 1)

#endif



/* Main functon
 *
 * Load boards from paths gives on the command-line,
 * solve them and print back to stdout.
 */

int main(int argc, char *argv[]) {

  int i, s;
  int ms = 10, pn = 5;
  params_t pars[pn];
  sudoku_t game;

  setparams(&pars[0], 5, "abcdefghijklmnopqrstuvwxy");
  setparams(&pars[1], 4, "0123456789abcdef");
  setparams(&pars[2], 4, "123456789abcdefg");
  setparams(&pars[3], 3, "123456789");
  setparams(&pars[4], 2, "1234");

  hipSetDeviceFlags(hipDeviceScheduleYield);

  for (i = 1; i < argc; i++) {

    if (i > 1) {
      putchar('\n');
      putchar('\n');
    }

    if (!load(&game, pars, pn, argv[i])) {
      fprintf(stderr, "Invalid input: %s\n", argv[i]);
      return 1;
    }

#ifndef BRUTEFORCE /* Deterministic solution */

    s = solve(&game);
    showmeta(&game, s);
    show(&game);
    
#else /* Deterministic + brute-force --> all solutions */

    /* Solve2 will print for us... */

    s = solve2(&game, ms);

    if (game.solutions >= ms) {
      fprintf(stderr, "\nBailing out after %d solutions.\n", ms);

    } else if (s < 0) {
      showmeta(&game, s);
      show(&game);
    }

#endif

    showmeta2(&game);
    
    unload(&game);
    
  }
  
  return 0;

}


void setparams(params_t *p, int dimension, const char *syms) {

  p->dimension = dimension;
  p->groupsize = p->dimension * p->dimension;
  p->boardsize = p->groupsize * p->groupsize;
  p->allbits = (1 << p->groupsize) - 1;
  p->symbols = syms;  
}


/* Loading and printing
 *
 * Errors cause the program to abort here,
 * so functions calling us do not need to do
 * error-checking. */

/* Load a board from file.
 * 
 * This simply reads groupsize optionally whitespace
 * -separated symbols from the given file. The input
 * may be formatted, but this is not a requirement.
 */

int load(sudoku_t *game, params_t *pars, int pn, char *path) {

  int i, p, fd, len;
  char buf[1024], *s;

  /* Open file */
  
  if ((fd = open(path, O_RDONLY)) == -1) {
    fprintf(stderr, "Could not open file %s: %m\n", path);
    exit(1);
  }

  /* Read data */
  
  if ((len = read(fd, buf, sizeof(buf))) == -1) {
    fprintf(stderr, "Could not read from file: %m\n");
    exit(1);
  }

  close(fd);

  game->board = NULL;


  /* Try params in order */

  for (p = 0; p < pn; p++) {
    
    /* Init parser */
  
    buf[len] = 0; /* null is not space and not a symbol, */
    s = buf;      /* so passing over it will always fail */

    /* Allocate board */
    
    if ((game->board = (field *) realloc(game->board, pars[p].boardsize * sizeof(field))) == NULL) {
      fprintf(stderr, "Could not allocate space for board: %m\n");
      exit(1);
    }


    /* Fill the board */
  
    for (i = 0; i < pars[p].boardsize; i++) {

      /* Skip spaces */
      while (*s == ' ' || *s == '\t' || *s == '\n')
	s++;

      /* Get one symbol */
      if (!sym_to_field(&pars[p], *s, &(game->board[i])))
	break;

      s++;

    }

    /* Check if all fields could be read */
    if (i == pars[p].boardsize) {
      break;
    }
    
  }

  /* Fail if we ran out of possible pars */
  if (p == pn) {
    free(game->board);
    return 0;
  }

  
  /* Store pars in game */

  game->pars = &pars[p];
  game->steps = 0;
  game->tries = 0;
  game->allsteps = 0;
  game->alltries = 0;
  game->solutions = 0;
  game->path = path;
#if 0
  game->pars->groups = NULL;
  game->pars->ms = NULL;

  if (cudaMalloc((void **)&(game->pars->groups), sizeof(field) * game->pars->boardsize * 3) != cudaSuccess ||
      cudaMalloc((void **)&(game->pars->ms), sizeof(int) * game->pars->groupsize * 3) != cudaSuccess) {
    fprintf(stderr, "Failed to allocate CUDA groups buffer\n");
    exit(1);
  }
#endif
#ifdef COMB_BUF
  game->pars->buf = NULL;

  if (cudaMalloc((void **)&game->pars->buf, (game->pars->allbits + 1) * sizeof(field) * game->pars->groupsize * 3) != cudaSuccess) {
    fprintf(stderr, "Failed to allocate CUDA combinations buffer\n");
    exit(1);
  }
#endif

  return 1;

}

void unload(sudoku_t *game) {
  free(game->board);
#ifdef COMB_BUF
  cudaFree(game->pars->buf);
#endif
}


/* Print solution information to standard output */
void showmeta(sudoku_t *game, int s) {

  if (s > 0) {

    printf("\nSudoku %s, solution %d: %s in %d steps, %d tries:\n\n",
	   game->path, game->solutions, statusmsg(s),
	   game->steps, game->tries);

  } else {

    printf("\nSudoku %s: %s after %d steps, %d tries:\n\n",
	   game->path, statusmsg(s),
	   game->steps, game->tries);
  }

}


/* Print total work information to standard output */
void showmeta2(sudoku_t *game) {
  printf("\nSudoku %s: found %d solution(s) in %d steps, %d tries\n",
	 game->path, game->solutions,
	 game->allsteps, game->alltries);
}

/* Print board to standard output.
 * 
 * This prints the fields as symbols, interspersed with
 * whitespace for formatting.
 */
void show(sudoku_t *game) {
  showboard(game->pars, game->board);
}

static void showboard(params_t *p, field *board) {

  int i;
  
  for (i = 0; i < p->boardsize; i++) {

    if (i == 0) {
    } else if (i % (p->groupsize * p->dimension) == 0) {
      putchar('\n');
      putchar('\n');
    } else if (i % p->groupsize == 0) {
      putchar('\n');
    } else if (i % p->dimension == 0) {
      putchar(' ');
      putchar(' ');
    } else {
      putchar(' ');
    }

    putchar(field_to_sym(p, board[i]));
    
  }

  putchar('\n');

}


/* Status message */

const char *statusmsg(int s) {

  if (s < 0)
    return "failed";
  
  if (s == 0)
    return "undefined";
  
  return "solved";

}


/* Symbol parsing and printing */

static int sym_to_field(params_t *p, char c, field *b) {

  field i;
  const char *s;
  
  if (c == '_' || c == '.') {
    *b = p->allbits;
    return 1;
  }
  
  if (c == '*') {
    *b = 0;
    return 1;
  }

  for (i = 1, s = p->symbols; *s; s++, i <<= 1)
    if (c == *s)
      break;

  if (*s) {
    *b = i;
    return 1;
  }

  return 0;

}

static char field_to_sym(params_t *p, field b) {

  if (b == 0)
    return '*';
  if (b == p->allbits)
    return '_';
  if (nbits(b) > 1)
    return '.';

  return p->symbols[fstbit(b)];

}


/* Resolution */

int solve(sudoku_t *game) {

  params_t *p = game->pars;
  field *board = game->board;
  int s;
  
  while (status(p, board) == 0 && solve1(p, board)) {
    game->steps++;
    game->allsteps++;
  }
  
  s = verify(p, board);

  if (s > 0)
    game->solutions++;

  return s;

}


/* Brute-force when solution got stuck
 * and print all possible solutions to
 * standard output.
 */

int solve2(sudoku_t *game, int ms) {

  params_t *p = game->pars;
  field *board = game->board;

  field b;
  int i, s;

  field *board_;
  int steps_, tries_;


  /* Try to solve deterministically */

  while (status(p, board) == 0 && solve1(p, board)) {
    game->steps++;
    game->allsteps++;
  }

  if ((s = verify(p, board)) != 0) {

    if (s > 0) {
      game->solutions++;
      showmeta(game, s);
      show(game);
    }

    return s;

  }


  /* Copy the original */

  if ((board_ = (field *) malloc(p->boardsize * sizeof(field))) == NULL) {
    fprintf(stderr, "Could not allocate memory for board: %m\n");
    exit(1);
  }

  memcpy(board_, board, p->boardsize * sizeof(field));
  steps_ = game->steps;
  tries_ = game->tries;


  /* Select first undetermined field */
  
  for (i = 0; i < p->boardsize; i++)
    if (nbits(board[i]) > 1)
      break;

  
  /* Try every possibility */

  b = board[i];
  while (b) {

    board[i] = 1 << fstbit(b);
    game->tries++;
    game->alltries++;
    clrbit(b);
    
    /* Recurse */

    solve2(game, ms);

    if (game->solutions >= ms)
      break;

    /* Revert to original */

    memcpy(board, board_, p->boardsize * sizeof(field));
    game->steps = steps_;
    game->tries = tries_;

  }


  /* Clean up */

  free(board_);
  return game->solutions ? 1 : -1;

}


/* One resolution step
 *
 * Solves rows, cols and blocks.
 * The return value indicates whether a change
 * was made (and another step might be helpful).
 * 
 * This function just does the copying around. The
 * actual resolution is done in solvegroup.
 */

static __constant__ const field groupsin[32*25*3] = {0};
static __device__ field groupsout[32*25*3];
static __device__ int msout[25 * 3];

static int solve1(params_t *p, field *board) {

  int i, j, m = 0;

  int ms[p->groupsize * 3];
  field groups[32 * p->groupsize * 3];

  field *rows   = &groups[0 * 32 * p->groupsize];
  field *cols   = &groups[1 * 32 * p->groupsize];
  field *blocks = &groups[2 * 32 * p->groupsize];

  hipError_t err;


  /* Extract cols and blocks */
  
  for (i = 0; i < p->groupsize; i++) {
    
    for (j = 0; j < p->groupsize; j++) {

      rows[i * 32 + j] = board[i * p->groupsize + j];
      cols[i * 32 + j] = board[j * p->groupsize + i];
      
      blocks[i * 32 + j] = board[  (i / p->dimension) * p->groupsize * p->dimension
				 + (i % p->dimension) * p->dimension
				 + (j / p->dimension) * p->groupsize
				 + (j % p->dimension)];
    }
    
  }

  
  /* Solve rows, cols, blocks */

  if ((err = hipMemcpyToSymbol(HIP_SYMBOL(groupsin), groups, sizeof(field) * p->groupsize * 32 * 3)) != hipSuccess) {
    fprintf(stderr, "Host to device copy failed: %s!\n", hipGetErrorString(err));
    exit(1);
  }

  solvegroup<<<3,p->groupsize>>>(p->boardsize, p->groupsize, p->allbits);

  if ((err = hipGetLastError()) != hipSuccess) {
    fprintf(stderr, "Kernel execution failed: %s!\n", hipGetErrorString(err));
    exit(1);    
  }

  if ((err = hipMemcpyFromSymbol(groups, HIP_SYMBOL(groupsout), sizeof(field) * p->groupsize * 32 * 3)) != hipSuccess ||
      (err = hipMemcpyFromSymbol(ms,     HIP_SYMBOL(msout),     sizeof(int) * p->groupsize * 3)) != hipSuccess) {
    fprintf(stderr, "Device to host copy failed: %s!\n", hipGetErrorString(err));
    exit(1);
  }


  /* Integrate results */
  
  for (i = 0; i < p->groupsize; i++) {

    for (j = 0; j < p->groupsize; j++) {

      board[i * p->groupsize + j] =

	rows[i * 32 + j] &
	cols[j * 32 + i] &
        blocks[  (i / p->dimension) * 32 * p->dimension
	       + (i % p->dimension) * p->dimension
	       + (j / p->dimension) * 32
	       + (j % p->dimension)];

    }
    
  }


  for (i = 0; i < p->groupsize * 3; i++)
    m += ms[i];

  
#ifdef DEBUG_STEP
  printf("\nStep (%d changes)\n\n", m);
  showboard(p, board);
  exit(0);
#endif

  return m;

}


/* Solve one group (row, column, block).
 *
 * For every field in the group, all values defined with
 * certainty in any of the combinations of the other
 * elements is set to zero.
 *
 * To do this, we loop over all possible combinations, denoted
 * by the bits in a counter from 1 to allbits, and save bitfields 
 * to a buffer marking the values that must be present in every
 * combination. This bitfield is found by or'ing all fields contained
 * in the combination, giving a list of all possible values in the
 * combination, and write this value if there are at most as many
 * possibilities as there are elements in the combination, or else zero.
 *
 * The values of the buffer can then be or'ed together for every index
 * for which the n'th bit is zero to get a bitfield of the defined values
 * in all combinations that do not include field n.
 */


static __global__ void solvegroup(int boardsize, int groupsize, int allbits) {

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const field *group = &groupsin[idx * 32];
  field *groupout = &groupsout[idx * 32];

  __shared__ field f[25];
  __shared__ field g[25];
  
  int m = 0;
  field b, c;


  g[0] = group[0];
  g[1] = group[1];
  g[2] = group[2];
  g[3] = group[3];
  g[4] = group[4];
  g[5] = group[5];
  g[6] = group[6];
  g[7] = group[7];
  g[8] = group[8];
  g[9] = group[9];
  g[10] = group[10];
  g[11] = group[11];
  g[12] = group[12];
  g[13] = group[13];
  g[14] = group[14];
  g[15] = group[15];
  g[16] = group[16];
  g[17] = group[17];
  g[18] = group[18];
  g[19] = group[19];
  g[20] = group[20];
  g[21] = group[21];
  g[22] = group[22];
  g[23] = group[23];
  g[24] = group[24];


  /* Calculate defined combinations */
  
  f[0] = 0;
  f[1] = 0;
  f[2] = 0;
  f[3] = 0;
  f[4] = 0;
  f[5] = 0;
  f[6] = 0;
  f[7] = 0;
  f[8] = 0;
  f[9] = 0;
  f[10] = 0;
  f[11] = 0;
  f[12] = 0;
  f[13] = 0;
  f[14] = 0;
  f[15] = 0;
  f[16] = 0;
  f[17] = 0;
  f[18] = 0;
  f[19] = 0;
  f[20] = 0;
  f[21] = 0;
  f[22] = 0;
  f[23] = 0;
  f[24] = 0;

  for (c = 1; c < allbits; c++) {

    b =
      ((c & (1 << 0)) ? g[0] : 0) |
      ((c & (1 << 1)) ? g[1] : 0) |
      ((c & (1 << 2)) ? g[2] : 0) |
      ((c & (1 << 3)) ? g[3] : 0) |
      ((c & (1 << 4)) ? g[4] : 0) |
      ((c & (1 << 5)) ? g[5] : 0) |
      ((c & (1 << 6)) ? g[6] : 0) |
      ((c & (1 << 7)) ? g[7] : 0) |
      ((c & (1 << 8)) ? g[8] : 0) |
      ((c & (1 << 9)) ? g[9] : 0) |
      ((c & (1 << 10)) ? g[10] : 0) |
      ((c & (1 << 11)) ? g[11] : 0) |
      ((c & (1 << 12)) ? g[12] : 0) |
      ((c & (1 << 13)) ? g[13] : 0) |
      ((c & (1 << 14)) ? g[14] : 0) |
      ((c & (1 << 15)) ? g[15] : 0) |
      ((c & (1 << 16)) ? g[16] : 0) |
      ((c & (1 << 17)) ? g[17] : 0) |
      ((c & (1 << 18)) ? g[18] : 0) |
      ((c & (1 << 19)) ? g[19] : 0) |
      ((c & (1 << 20)) ? g[20] : 0) |
      ((c & (1 << 21)) ? g[21] : 0) |
      ((c & (1 << 22)) ? g[22] : 0) |
      ((c & (1 << 23)) ? g[23] : 0) |
      ((c & (1 << 24)) ? g[24] : 0);

    if (__popc(c) < __popc(b))
      continue;

    f[0] |= (c & (1 << 0)) ? 0 : b;
    f[1] |= (c & (1 << 1)) ? 0 : b;
    f[2] |= (c & (1 << 2)) ? 0 : b;
    f[3] |= (c & (1 << 3)) ? 0 : b;
    f[4] |= (c & (1 << 4)) ? 0 : b;
    f[5] |= (c & (1 << 5)) ? 0 : b;
    f[6] |= (c & (1 << 6)) ? 0 : b;
    f[7] |= (c & (1 << 7)) ? 0 : b;
    f[8] |= (c & (1 << 8)) ? 0 : b;
    f[9] |= (c & (1 << 9)) ? 0 : b;
    f[10] |= (c & (1 << 10)) ? 0 : b;
    f[11] |= (c & (1 << 11)) ? 0 : b;
    f[12] |= (c & (1 << 12)) ? 0 : b;
    f[13] |= (c & (1 << 13)) ? 0 : b;
    f[14] |= (c & (1 << 14)) ? 0 : b;
    f[15] |= (c & (1 << 15)) ? 0 : b;
    f[16] |= (c & (1 << 16)) ? 0 : b;
    f[17] |= (c & (1 << 17)) ? 0 : b;
    f[18] |= (c & (1 << 18)) ? 0 : b;
    f[19] |= (c & (1 << 19)) ? 0 : b;
    f[20] |= (c & (1 << 20)) ? 0 : b;
    f[21] |= (c & (1 << 21)) ? 0 : b;
    f[22] |= (c & (1 << 22)) ? 0 : b;
    f[23] |= (c & (1 << 23)) ? 0 : b;
    f[24] |= (c & (1 << 24)) ? 0 : b;

  }

  
  /* Resolve group */
  
  groupout[0] = g[0] & (allbits ^ f[0]);
  groupout[1] = g[1] & (allbits ^ f[1]);
  groupout[2] = g[2] & (allbits ^ f[2]);
  groupout[3] = g[3] & (allbits ^ f[3]);
  groupout[4] = g[4] & (allbits ^ f[4]);
  groupout[5] = g[5] & (allbits ^ f[5]);
  groupout[6] = g[6] & (allbits ^ f[6]);
  groupout[7] = g[7] & (allbits ^ f[7]);
  groupout[8] = g[8] & (allbits ^ f[8]);
  groupout[9] = g[9] & (allbits ^ f[9]);
  groupout[10] = g[10] & (allbits ^ f[10]);
  groupout[11] = g[11] & (allbits ^ f[11]);
  groupout[12] = g[12] & (allbits ^ f[12]);
  groupout[13] = g[13] & (allbits ^ f[13]);
  groupout[14] = g[14] & (allbits ^ f[14]);
  groupout[15] = g[15] & (allbits ^ f[15]);
  groupout[16] = g[16] & (allbits ^ f[16]);
  groupout[17] = g[17] & (allbits ^ f[17]);
  groupout[18] = g[18] & (allbits ^ f[18]);
  groupout[19] = g[19] & (allbits ^ f[19]);
  groupout[20] = g[20] & (allbits ^ f[20]);
  groupout[21] = g[21] & (allbits ^ f[21]);
  groupout[22] = g[22] & (allbits ^ f[22]);
  groupout[23] = g[23] & (allbits ^ f[23]);
  groupout[24] = g[24] & (allbits ^ f[24]);

  m =
    (groupout[0] == g[0] ? 0 : 1) + 
    (groupout[1] == g[1] ? 0 : 1) + 
    (groupout[2] == g[2] ? 0 : 1) + 
    (groupout[3] == g[3] ? 0 : 1) +
    (groupout[4] == g[4] ? 0 : 1) + 
    (groupout[5] == g[5] ? 0 : 1) + 
    (groupout[6] == g[6] ? 0 : 1) + 
    (groupout[7] == g[7] ? 0 : 1) + 
    (groupout[8] == g[8] ? 0 : 1) + 
    (groupout[9] == g[9] ? 0 : 1) +
    (groupout[10] == g[10] ? 0 : 1) + 
    (groupout[11] == g[11] ? 0 : 1) + 
    (groupout[12] == g[12] ? 0 : 1) + 
    (groupout[13] == g[13] ? 0 : 1) + 
    (groupout[14] == g[14] ? 0 : 1) + 
    (groupout[15] == g[15] ? 0 : 1) +
    (groupout[16] == g[16] ? 0 : 1) + 
    (groupout[17] == g[17] ? 0 : 1) + 
    (groupout[18] == g[18] ? 0 : 1) + 
    (groupout[19] == g[19] ? 0 : 1) + 
    (groupout[20] == g[20] ? 0 : 1) + 
    (groupout[21] == g[21] ? 0 : 1) + 
    (groupout[22] == g[22] ? 0 : 1) + 
    (groupout[23] == g[23] ? 0 : 1) + 
    (groupout[24] == g[24] ? 0 : 1);

  msout[idx] = m;

}

#ifndef CPUEXT

static inline int nbits(field b) {

  int n = 0;
  
  while (b) {
    n += b & 1;
    b >>= 1;
  }

  return n;
  
}

static inline int fstbit(field b) {

  int i = 0;

  if (b == 0)
    return -1;
  
  while ((b & 1) == 0) {
    i++;
    b >>= 1;
  }
  
  return i;

}

#endif


static __device__ inline field combination(const field *group, field c) {

  field a = 0;

  while (c) {
    a |= group[__ffs(c) - 1];
    clrbit(c);
  }
  
  return a;

}


/* Verify status of board:
 * if one value is 0, status is failed (-1)
 * if one value has more than 1 bit set, status is undefined (0)
 * if all values have exactly 1 bit set, status is solved (1)
 */

static int status(params_t *p, field *board) {

  int i;

  /* Check for failure */
  for (i = 0; i < p->boardsize; i++)
    if (board[i] == 0)
      return -1;

  /* Check for undef */
  for (i = 0; i < p->boardsize; i++)
    if (nbits(board[i]) > 1)
      return 0;

  return 1;

}


/* Verify correctness */

static int verify(params_t *p, field *board) {

  field a;
  int i, j, s;

  
  s = status(p, board);

  if (s < 0)
    return s;


  /* Check rows */

  for (i = 0; i < p->groupsize; i++) {
    
    for (j = 0, a = 0; j < p->groupsize; j++)
      a |= board[i * p->groupsize + j];
    
    if (a != p->allbits)
      return -1;
  }

  
  /* Check columns */

  for (i = 0; i < p->groupsize; i++) {
    
    for (j = 0, a = 0; j < p->groupsize; j++)
      a |= board[j * p->groupsize + i];
    
    if (a != p->allbits)
      return -1;
  }


  /* Check blocks */

  for (i = 0; i < p->groupsize; i++) {
    
    for (j = 0, a = 0; j < p->groupsize; j++)
      a |= board[  (i / p->dimension) * p->groupsize * p->dimension
		 + (i % p->dimension) * p->dimension
		 + (j / p->dimension) * p->groupsize
		 + (j % p->dimension)];
    
    if (a != p->allbits)
      return -1;
  }

  return s;
  
}
